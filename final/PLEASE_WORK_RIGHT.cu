#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<cmath>
#define TILE_SIZE 4            // Tile size and block size, both are taken as 32
__device__ void store_full_row(float*,float*,int,int, int, int);
__device__ void load_full_row(float*,float*,int,int, int, int);
__device__ void store_full(float*,float*,int,int,int, int, int);
__device__ void load_full(float*,float*,int,int,int, int, int);
__device__ void store_lower(float*,float*,int,int,int, int, int);
__device__ void load_lower(float*,float*,int,int,int, int, int);
__device__ void potrf_tile(float*);
__device__ void trsm_tile(float*,int,int,int);
__device__ void syrk_tile(float*,float*,int,int,int);
__global__ void right_looking_launch_kernel(float*,int);
__device__ void store_zeros(float*,int);

__device__ void store_full_row(float* read_data,float* write_data,int i,int N, int M, int shared_size_single_matrix)
{
    int global_y;
    int global_x = i*blockDim.y + threadIdx.y;
    for(int j=0;j<N/TILE_SIZE;j++)
    {
        global_y = j*blockDim.z + threadIdx.z;
        write_data[global_y*N*M + global_x*M +  blockIdx.x * blockDim.x + threadIdx.x] = read_data[threadIdx.y + (TILE_SIZE+1)*global_y + threadIdx.x*shared_size_single_matrix];
    }
    __syncthreads();
}
__device__ void load_full_row(float* read_data,float* write_data,int i,int N, int M, int shared_size_single_matrix)
{
    int global_y;
    int global_x = i*blockDim.y + threadIdx.y;
    for(int j=0;j<N/TILE_SIZE;j++)
    {
        global_y = j*blockDim.z + threadIdx.z;
        write_data[threadIdx.y + (TILE_SIZE+1)*global_y + threadIdx.x*shared_size_single_matrix] = read_data[global_y*N*M + global_x*M +  blockIdx.x * blockDim.x + threadIdx.x];
        // printf("%d, %d\n", threadIdx.y + (TILE_SIZE+1)*global_y + threadIdx.x*shared_size_single_matrix, global_y*N*M + global_x*M + threadIdx.x);
    }
    __syncthreads();
}
__device__ void store_full(float* read_data,float* write_data,int i,int j,int N, int M, int shared_size_single_matrix)
{
    int global_y = j*blockDim.z + threadIdx.z;
    int global_x = i*blockDim.y + threadIdx.y;
    write_data[global_y*N*M + global_x*M +  blockIdx.x * blockDim.x + threadIdx.x] = read_data[threadIdx.y + (TILE_SIZE+1)*threadIdx.z + threadIdx.x*shared_size_single_matrix];
    __syncthreads();
}
__device__ void load_full(float* read_data,float* write_data,int i,int j,int N, int M, int shared_size_single_matrix)
{
    int global_y = j*blockDim.z + threadIdx.z;
    int global_x = i*blockDim.y + threadIdx.y;
    write_data[threadIdx.y + (TILE_SIZE+1)*threadIdx.z + threadIdx.x*shared_size_single_matrix] = read_data[global_y*N*M + global_x*M +  blockIdx.x * blockDim.x + threadIdx.x];
    __syncthreads();
}
__device__ void store_lower(float* read_data,float* write_data,int i,int j,int N, int M, int shared_size_single_matrix)
{
    int global_y = j*blockDim.z + threadIdx.z;
    int global_x = i*blockDim.y + threadIdx.y;
    // printf("%f is at %d\n", read_data[threadIdx.y + (TILE_SIZE+1)*threadIdx.z + threadIdx.x*shared_size_single_matrix], threadIdx.y + (TILE_SIZE+1)*threadIdx.z + threadIdx.x*shared_size_single_matrix);
    if(threadIdx.z >= threadIdx.y)
        write_data[global_y*N*M + global_x*M +  blockIdx.x * blockDim.x + threadIdx.x] = read_data[threadIdx.y + (TILE_SIZE+1)*threadIdx.z + threadIdx.x*shared_size_single_matrix];
    else
        write_data[global_y*N*M + global_x*M +  blockIdx.x * blockDim.x + threadIdx.x] = 0.0;
    __syncthreads();
}
__device__ void load_lower(float* read_data,float* write_data,int i,int j,int N, int M, int shared_size_single_matrix)
{
    int global_y = j*blockDim.z + threadIdx.z;
    int global_x = i*blockDim.y + threadIdx.y;
    if(threadIdx.z >= threadIdx.y)
        write_data[threadIdx.y + (TILE_SIZE+1)*threadIdx.z + threadIdx.x*shared_size_single_matrix] = read_data[global_y*N*M + global_x*M +  blockIdx.x * blockDim.x + threadIdx.x];
    else
        write_data[threadIdx.y + (TILE_SIZE+1)*threadIdx.z + threadIdx.x*shared_size_single_matrix] = 0.0;
    __syncthreads();
}
__device__ void potrf_tile(float* t_A)
{
    int t_x = threadIdx.y;
    int t_y = threadIdx.z;
    __shared__ float temp2;
    for(int k=0;k<TILE_SIZE;k++)
    {
        if(t_x==t_y && t_x==k)
        {
            t_A[k*(TILE_SIZE+1) + k] = sqrtf(t_A[k*(TILE_SIZE+1) + k]);
            temp2 = t_A[k*(TILE_SIZE+1) + k];
        }
        __syncthreads();
        if(t_x<t_y && t_x == k)
        {
            t_A[t_y*(TILE_SIZE+1) + k]/= temp2;
        }
        __syncthreads();
        if(k<t_y && k<t_x && t_x<=t_y)
        {
            t_A[t_y*(TILE_SIZE+1) + t_x]-= t_A[t_x*(TILE_SIZE+1) + k]*t_A[t_y*(TILE_SIZE+1) + k];
        }
        __syncthreads();
    }
}
__device__ void trsm_tile(float *row_data,int i,int j,int N)
{
    int global_y = j*blockDim.z + threadIdx.z;
    int global_x = i*blockDim.y + threadIdx.y;
    int t_x = threadIdx.y;
    int t_y = threadIdx.z;
    for(int s=0;s<TILE_SIZE;s++)
    {
	if(t_x==s)
        {
	    row_data[global_y*(TILE_SIZE+1) + t_x]/= row_data[global_x*(TILE_SIZE+1) + t_x];
	}
	__syncthreads();
	if(t_x > s)
        {
	    row_data[global_y*(TILE_SIZE+1) + t_x]-= row_data[global_x*(TILE_SIZE+1) +  s]*row_data[global_y*(TILE_SIZE+1) + s];
	}
	__syncthreads();
    }
}
__device__ void syrk_tile(float* row_data,float* edit_data,int i,int j,int N) 
{
    int global_y = j*blockDim.z + threadIdx.z;
    int global_x = i*blockDim.y + threadIdx.y;
    int t_y = threadIdx.z;
    int t_x = threadIdx.y;
    float valueToSubtract = 0.0;
    for(int r=0;r<TILE_SIZE;r++)
    {
        valueToSubtract+= row_data[r + global_y*(TILE_SIZE+1)]*row_data[r + global_x*(TILE_SIZE+1)];
    }
    edit_data[t_y*(TILE_SIZE+1) + t_x]-= valueToSubtract;
    __syncthreads();
}

__device__ void store_zeros(float* write_data,int N, int M)
{
    int t_y = threadIdx.z;
    int t_x = threadIdx.y;
    int i,j;
    for(i=0;i<N/TILE_SIZE-1;i++)
    {
        for(j=i+1;j<N/TILE_SIZE;j++)
        {
            int global_x = j*blockDim.z + threadIdx.z;
            int global_y = i*blockDim.y + threadIdx.y;
            write_data[global_y*N*M + global_x*M + blockIdx.x * blockDim.x +threadIdx.x]  = 0.0;

        }
            // A[j*blockDim.x + t_x + (i*blockDim.y + t_y)*N] = 0.0;
    }
    __syncthreads();
}

__global__ void right_looking_launch_kernel(float* read_data,int N, int M , int num_of_matrices_per_block, int shared_size_single_matrix) // N -> dim, M -> num of matrices per block
{

    int no_of_tiles = (N / TILE_SIZE) + (N % TILE_SIZE != 0); 
    
    int tx = threadIdx.x;
    float *rA1 = NULL;

    extern __shared__ float row_data[];
    // __shared__ float tile_data[TILE_SIZE*(TILE_SIZE+1)];                // Using TILE_SIZE+1 to avoid Band-conflict in Shared Memory
    int tile_data_index = M * (N*(TILE_SIZE+1) + TILE_SIZE*(TILE_SIZE+1) + 1);
    // __shared__ float* tile_data = &row_data[M * (N*(TILE_SIZE+1) + TILE_SIZE*(TILE_SIZE+1) + 1)];
    int shared_size_single_matrix_tile_data = TILE_SIZE * (TILE_SIZE + 1);


    int i,j,k;
    for(i=0;i<N/TILE_SIZE;i++)
    {
        load_lower(read_data,&row_data[tile_data_index],i,i,N, M, shared_size_single_matrix_tile_data);
        // printf("%d \n", tile_data_index + shared_size_single_matrix_tile_data * M);
        // if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            // for (int z = tile_data_index; z < tile_data_index + shared_size_single_matrix_tile_data * M; z++) {
                // printf("%f is at %d\n", row_data[z], z);
            // }
        // }
        
        rA1 = &row_data[tile_data_index + tx*shared_size_single_matrix_tile_data];
        // printf("%d\n", tx*shared_size_single_matrix_tile_data);
        // potrf_tile(tile_data);
        potrf_tile(rA1);
        store_lower(&row_data[tile_data_index],read_data,i,i,N, M, shared_size_single_matrix_tile_data);
        load_full_row(read_data,row_data,i,N, M, shared_size_single_matrix);
        for(j=i+1;j<N/TILE_SIZE;j++)
        {
            trsm_tile(&row_data[tx*shared_size_single_matrix],i,j,N);
            for(k=i+1;k<j;k++)
            {
                load_full(read_data,&row_data[tile_data_index],k,j,N, M, shared_size_single_matrix_tile_data);
                rA1 = &row_data[tile_data_index + tx*shared_size_single_matrix_tile_data];
                // syrk_tile(row_data,tile_data,k,j,N);
                syrk_tile(&row_data[tx*shared_size_single_matrix],rA1,k,j,N);
                store_full(&row_data[tile_data_index],read_data,k,j,N, M, shared_size_single_matrix_tile_data);
            }
            load_full(read_data,&row_data[tile_data_index],k,j,N, M, shared_size_single_matrix_tile_data);
            syrk_tile(&row_data[tx*shared_size_single_matrix],&row_data[tile_data_index + tx*shared_size_single_matrix_tile_data],k,j,N);
            store_full(&row_data[tile_data_index],read_data,k,j,N, M, shared_size_single_matrix_tile_data);
        }
        store_full_row(row_data,read_data,i,N, M, shared_size_single_matrix);
    }
    store_zeros(read_data,N,M);
}


int main()
{
    // int n,N;
    // printf("Enter dimension (N) : ");
    // scanf("%d",&n);
    // if((n%TILE_SIZE)==0)
    //     N = n;
    // else
    //     N = (((int) (n/TILE_SIZE)) + 1)*TILE_SIZE;
    // size_t size = N*N*sizeof(float);
    // float *M = (float *)malloc(size);
    // if(M == NULL)
    // {
    //     fprintf(stderr,"Failed to allocate host vectors!\n");
    //     exit(EXIT_FAILURE);
    // }
    // int i,j;
    // printf("Enter input matrix: \n");
    // for(i=0;i<N;i++)
    // {
    //     for(j=0;j<N;j++)
    //     {
    //         if(i>=n || j>=n)
    //             M[i*N + j] = 1;     //Padding the matrix with 1
    //         else
    //             scanf("%f",&M[i*N + j]);
    //     }
    // }

    FILE *fptr;
    fptr = fopen("./dataset/size4_256matrices.txt", "r");
    int num_of_matrices, dim_of_matrix;
    fscanf(fptr, "%d", &num_of_matrices);
    fscanf(fptr, "%d", &dim_of_matrix);
    float read_element;
    float* h_A = NULL;
    int numElements = num_of_matrices * dim_of_matrix * dim_of_matrix;
    size_t size = numElements * sizeof(float);
    hipDeviceProp_t devp;
    hipGetDeviceProperties(&devp, 0);

    h_A = (float *)malloc(size);
    
    int global_id = 0;

    for (int matrix_index = 0; matrix_index < num_of_matrices; matrix_index++)
    {
        for (int row = 0; row < dim_of_matrix; row++)
        {
            for (int column = 0; column < dim_of_matrix; column++)
            {
                fscanf(fptr, "%f", &read_element);
                global_id = row * dim_of_matrix * num_of_matrices + column * num_of_matrices + matrix_index;
                h_A[global_id] = read_element;
                // printf("At pos %d we get %0.2f\n", global_id, h_A[global_id]);
                // printf("%0.2f \n ", h_A[global_id]);
            }
        }
    }
    printf("\nRead from the input file successfully!\n");
    fclose(fptr);

    printf("\nPrinting the host-side input array read from the input file:\n");
    for (int i = 0; i < numElements; i++) {    
        printf("%f ", h_A[i]);
    }
    printf("\n\n");



    // hipError_t err = hipSuccess;
    // float *read_data = NULL;
    // err = hipMalloc((void **)&read_data,N*N*sizeof(float));
    // if(err != hipSuccess)
    // {
    //     fprintf(stderr,"Failed to allocate matrix on the CUDA device! (error code %s)\n",hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // printf("Coping the matrix from host memory to device memory\n");
    // err = hipMemcpy(read_data,M,size,hipMemcpyHostToDevice);
    // if(err != hipSuccess)
    // {
    //     fprintf(stderr,"Failed to copy matrix from host to device (error code %s)\n",hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // printf("Testing for matrix M [%dx%d]\n",N,N);

    hipError_t err = hipSuccess;

    float *d_A = NULL;

    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else {
        printf("Copied the h_A to device side successfully!\n\n");
    }



    // dim3 grid(1,1,1);
    // dim3 block(TILE_SIZE,TILE_SIZE,1);
    // size_t shared_size = (N*(TILE_SIZE+1) + TILE_SIZE*(TILE_SIZE+1) + 1)*sizeof(float);
    // right_looking_launch_kernel<<<grid,block,shared_size>>>(read_data,N);
    // err = hipMemcpy(M,read_data,size,hipMemcpyDeviceToHost);
    // if(err != hipSuccess)
    // {
    //     fprintf(stderr, "Failed to copy the output matrix M from device to Host (error code %s)\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    
    // int num_of_matrices_per_block = num_of_matrices;
    int num_of_matrices_per_block = min(128/(TILE_SIZE * TILE_SIZE) , num_of_matrices);	
    dim3 grid((num_of_matrices) / num_of_matrices_per_block , 1, 1);	
    dim3 block(num_of_matrices_per_block, TILE_SIZE, TILE_SIZE);

    // dim3 grid(1, 1, 1);
    // dim3 block(num_of_matrices, TILE_SIZE, TILE_SIZE);
    // no of tiles in a column
    // int INPUT_SIZE = dim_of_matrix;
    // int no_of_tiles = (INPUT_SIZE / TILE_SIZE) + (INPUT_SIZE % TILE_SIZE != 0); // ceil of (INPUT_SIZE / TILE_SIZE)
    int N = dim_of_matrix;
    size_t shared_size = num_of_matrices * (N*(TILE_SIZE+1) + TILE_SIZE*(TILE_SIZE+1) + 1)*sizeof(float) + num_of_matrices_per_block * TILE_SIZE*(TILE_SIZE+1) * sizeof(float);
    
    right_looking_launch_kernel<<<grid,block,shared_size>>>(d_A, dim_of_matrix, num_of_matrices, num_of_matrices ,(num_of_matrices * (N*(TILE_SIZE+1) + TILE_SIZE*(TILE_SIZE+1) + 1))/num_of_matrices);
    //left_looking_kernel<<<grid, block, num_of_matrices_per_block * 1 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_A, dim_of_matrix, num_of_matrices ,1 * TILE_SIZE * TILE_SIZE);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n",
                hipGetErrorString(cudaerr));
    }

    // if(TILE_SIZE == INPUT_SIZE)
    // {
    //     // printf("The if statement works.\n");
    //     left_looking_kernel<<<grid, block, num_of_matrices * 1 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_A, dim_of_matrix, num_of_matrices ,1 * TILE_SIZE * TILE_SIZE);
    // }

    // else if((no_of_tiles + 2) * TILE_SIZE * TILE_SIZE * sizeof(float) < devp.sharedMemPerBlock)
    // {
    //     //printf("The if statement works.\n");
    //     left_looking_kernel_less_mem<<<grid, block, num_of_matrices * 4 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_A, dim_of_matrix, num_of_matrices ,4 * TILE_SIZE * TILE_SIZE);
    //     // left_looking_kernel<<<grid, block,num_of_matrices * (no_of_tiles + 2) * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_A, dim_of_matrix, num_of_matrices ,(no_of_tiles + 2) * TILE_SIZE * TILE_SIZE);
    // }
    // else
    // {
    //     left_looking_kernel_less_mem<<<grid, block, num_of_matrices * 4 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_A, dim_of_matrix, num_of_matrices ,4 * TILE_SIZE * TILE_SIZE);
    // }






    // printf("Printing output matrix\n");
    // for(i=0;i<n;i++)
    // {
    //     for(j=0;j<n;j++)
    //     {
    //         if(j<=i)
    //             printf("%f\t",M[i*N + j]);
    //         else
    //             printf("%f\t",0.0);
    //     }
    //     printf("\n");
    // }
    // err = hipFree(read_data);
    // if(err != hipSuccess)
    // {
    //     fprintf(stderr, "Failed to free device matrix M (error code %s)\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // err = hipDeviceReset();
    // if(err != hipSuccess)
    // {
    //     fprintf(stderr, "Failed to deinitialize the CUDA device (error code %s)\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // free(M);
    // printf("DONE!\n");

    err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else {
        printf("\nCopied d_A to host side successfully!\n");
    }
    
    printf("\nPrinting the output of cudememcopyDeviceToHost, i.e. the host-side array returned from device side:\n");
    for (int i = 0; i < numElements; i++) {    
        printf("%f ", h_A[i]);
    }


    err = hipFree(d_A);
    if(err != hipSuccess)
    {
        fprintf(stderr, "\nFailed to free device matrix M (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceReset();
    if(err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the CUDA device (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    FILE *fptr1;
    fptr1 = fopen("./output_r.txt", "w+");
    float write_element;
    fprintf(fptr1, "%d\n", num_of_matrices);
    fprintf(fptr1, "%d\n", dim_of_matrix);

    for (int matrix_index = 0; matrix_index < num_of_matrices; matrix_index++)
    {
        for (int row = 0; row < dim_of_matrix; row++)
        {
            for (int column = 0; column < dim_of_matrix; column++)
            {
                //write_element = h_A[matrix_index * dim_of_matrix * dim_of_matrix + row * dim_of_matrix + column];
                global_id = row * dim_of_matrix * num_of_matrices + column * num_of_matrices + matrix_index;
                write_element = h_A[global_id] ;
                fprintf(fptr1, "%0.2f ", write_element);
            }
         fprintf(fptr1,"\n");
        }
        fprintf(fptr1,"\n");
    }
    fclose(fptr1);
    free(h_A);
    printf("\n\nAll tasks completed successfully!\n\n");

    return 0;
}
